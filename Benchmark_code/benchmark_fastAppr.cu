//%%writefile benchmark_fastAppr.cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Check CUDA error
#define CHECK_CUDA(call)                                          \
    {                                                             \
        hipError_t err = call;                                   \
        if (err != hipSuccess)                                   \
        {                                                         \
            printf("%s : %d", hipGetErrorString(err), __LINE__); \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    }

#define BLOCK_SIZE 8

// Kernels CUDA per le operazioni aritmetiche element-wise
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, const int size);
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, const int size);

__global__ void divide_matrices(const float* A, const float* B, float* C, const int size);
__global__ void multiply_matrices(const float* A, const float* B, float* C, const int size);

// Using cuda kernels to compute the DCT and the IDCT
// Those FUNC use intenal dct/idct kernel function, differs according to implementation used.
void dct_all_blocks_cuda(float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result);
void idct_all_blocks_cuda(const float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result);

int main(int argc, char *argv[])
{
    if (argc != 2) {
        printf("Use: %s <width/height>\n", argv[0]);
        return 1;
    }
    size_t input = strtoul(argv[1], NULL, 10);
    size_t width, height;
    width = input;
    height = input;

    float* image_matrix_float;
    image_matrix_float = (float*)malloc(width * height * sizeof(float));
    srand(42);
    for (int i = 0; i < height;i++) {
        for (int j = 0; j < width; j++) {
            image_matrix_float[i * width + j] = rand() % 256;;
        }
    }

    // Transform matrix (hardcoded for simplicity)
    float transform_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339,
            0.5, 0.5, 0, 0, 0, 0, -0.5, -0.5,
            0.4472136, 0.2236068, -0.2236068, -0.4472136, -0.4472136, -0.2236068, 0.2236068, 0.4472136,
            0, 0, -0.70710678, 0, 0, 0.70710678, 0, 0,
            0.35355339, -0.35355339, -0.35355339, 0.35355339, 0.35355339, -0.35355339, -0.35355339, 0.35355339,
            0.5, -0.5, 0, 0, 0, 0, 0.5, -0.5,
            0.2236068, -0.4472136, 0.4472136, -0.2236068, -0.2236068, 0.4472136, -0.4472136, 0.2236068,
            0, 0, 0, -0.70710678, 0.70710678, 0, 0, 0};

    // allocate host memory for the result image
    float *result;
    result = (float *)malloc(width * height * sizeof(float));

    // allocate device memory for: image_block (d_A), trasform_matrix (d_B), and result (d_C)
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, width * height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, width * height * sizeof(float)));

    // copy host memory to device
    CHECK_CUDA(hipMemcpy(d_A, image_matrix_float, width * height * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, transform_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Compute DCT
    // d_A = image_block ; d_B = transform_matrix ; d_C = result
    dct_all_blocks_cuda(d_A, height,width,d_B, d_C);

    // copy device memory to host
    // result = d_C
    CHECK_CUDA(hipMemcpy(result, d_C, width * height * sizeof(float), hipMemcpyDeviceToHost));

    // d_E = result of the idct applied on the block_image compressed
    float *d_E;
    CHECK_CUDA(hipMalloc(&d_E, width * height * sizeof(float)));

    // Compute the idct
    // d_C = result ; d_B = transform_matrix ; d_E = result
    idct_all_blocks_cuda(d_C, height, width, d_B,d_E);

    // copy device memory to host
    // result = d_E
    CHECK_CUDA(hipMemcpy(result, d_E, width * height * sizeof(float), hipMemcpyDeviceToHost));

    // Cleanup
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_E));
    free(result);
    free(image_matrix_float);
    return 0;
}

// Kernel CUDA per la sottrazione element-wise matrice - scalare
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const int Id_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int global = Id_y * gridDim.x * BLOCK_SIZE + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] - scalar;
    }
}

// Kernel CUDA per l'addizione element-wise matrice - scalare
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const int Id_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int global = Id_y * gridDim.x * BLOCK_SIZE + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] + scalar;
    }
}

// Kernel CUDA per la divisione elemento per elemento
__global__ void divide_matrices(const float* A, const float* B, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const int Id_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int global = Id_y * gridDim.x * BLOCK_SIZE + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  round(A[global] / B[threadIdx.y * BLOCK_SIZE + threadIdx.x]);
    }
}

// Kernel CUDA per la moltiplicazione elemento per elemento
__global__ void multiply_matrices(const float* A, const float* B, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const int Id_y = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    const int global = Id_y * gridDim.x * BLOCK_SIZE + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] * B[threadIdx.y * BLOCK_SIZE + threadIdx.x];
    }
}

/* *
 * Effettua la DCT utilizzando la matrice di trasformazione
 * (TRANSFORM_MATRIX @ IMAGE) @ TRANSFORM_MATRIX.T
 * La matrice di trasformazione è 8x8
 * shared_matrix = TRANSFORM_MATRIX @ IMAGE
 * result = shared_matrix @ TRANSFORM_MATRIX.T
 * */
__global__ void cuda_matrix_dct_paper(const float* image_matrix, const int img_size, const float* transform_matrix, float* result) {
    float riga[BLOCK_SIZE];
    __shared__ float shared_transform[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float shared_image[BLOCK_SIZE*BLOCK_SIZE*BLOCK_SIZE];
    /* *
     * shared_image logicamente divisa come un blocco 8x8 su una sola riga
     * Sono 8 blocchi 8x8, 8 righe da 64 float.
     * [[image_block_0]
     *  [image_block_i]
     *  [image_block_7]]
     * */
    // CUDA related vars (ids)
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;
    // Image size related vars ()
    int imgDimX = img_size / blockDim.x; // numero di blocchi 8x8 sull asse X
    int imgDimY = img_size / blockDim.y; // numero di blocchi 8x8 sull'asse Y
    int imageIdY = global / BLOCK_SIZE / imgDimX; // indicizzazione del blocco, Y
    int imageIdX = global / BLOCK_SIZE % imgDimY; // indicizzazione del blocco, X
    int offset_y = imageIdY * BLOCK_SIZE * img_size; // si "sposta" verso il basso di (BLOCK_SIZE * img_size)
    int offset_x = imageIdX * BLOCK_SIZE; // si "sposta" verso destra di BLOCK_SIZE

    float sums = 0;

    shared_transform[threadIdx.y * blockDim.x + threadIdx.x] = transform_matrix[threadIdx.y * blockDim.x + threadIdx.x];
    for(int i = 0;i < BLOCK_SIZE;i++)
        shared_image[threadIdx.y*(BLOCK_SIZE*BLOCK_SIZE) + (threadIdx.x*BLOCK_SIZE) + i] = image_matrix[(offset_y+offset_x) + (threadIdx.x*img_size) + i];
        //shared_image[VA CAMBIATO] = image_matrix[(offset_y) + (threadIdx.y*img_size) + (threadIdx.x*BLOCK_SIZE) + i];
    __syncthreads();
    /* *
     * Il seguente IF serve per evitare accessi illegali alla memoria.
     * Essendo che adesso i Threads sono mappati sul blocco immagine,
     * potrebbero esserci thread che non svolgono lavoro.
     * In tal caso bisogna evitare che accedano alla memoria.
     * */
    if (global >= imgDimX * imgDimY * BLOCK_SIZE)return;

    // RIGHE DI T PER COLONNE DI IMG (T @ X)
    for (int i = 0;i < BLOCK_SIZE;i++) {
        for (int j = 0;j < BLOCK_SIZE;j++) {
            // sums += T [ sempre la stessa riga ] * X [ colonne in sequenza ]
            //sums += shared_transform[threadIdx.x * BLOCK_SIZE + j] * image_matrix[i + (offset_y + offset_x) + (j * img_size)];
            sums += shared_transform[threadIdx.x * BLOCK_SIZE + j] * shared_image[threadIdx.y*(BLOCK_SIZE*BLOCK_SIZE) + i + j*BLOCK_SIZE];
        }
        // TX [ riga ] = T[ riga ] * X[ colonne ] (TX[riga] = somma dei prodotti)
        // shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + i] = sums;
        riga[i] = sums;
        sums = 0;
    }

    sums = 0;

    // RIGHE DI TX PER RIGHE DI T (TX @ T.T)
    for (int i = 0;i < BLOCK_SIZE;i++) {
        for (int j = 0;j < BLOCK_SIZE;j++) {
            // sums += TX [ sempre la stessa riga ] * T [ righe in sequenza ]
            // sums += shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + j] * transform_matrix[i * 8 + j];
            sums+= riga[j] * shared_transform[i * BLOCK_SIZE + j];
        }
        // result [ riga ] = TX [ riga ] * T [ righe ]
        result[(offset_y + offset_x) + (threadIdx.x * img_size) + i] = sums;
        sums = 0;
    }
}

/* *
 * Effettua la IDCT utilizzando la matrice di trasformazione
 * (TRANSFORM_MATRIX.T @ DCT_MATRIX) @ TRANSFORM_MATRIX
 * La matrice di trasformazione è 8x8
 * shared_matrix = TRANSFORM_MATRIX.T @ DCT_MATRIX
 * result = shared_matrix @ TRANSFORM_MATRIX
 * */
__global__ void cuda_matrix_idct_paper(const float* image_matrix, const int img_size,const float* transform_matrix, float* result) {
    float riga[BLOCK_SIZE];
    __shared__ float shared_transform[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float shared_image[BLOCK_SIZE*BLOCK_SIZE*BLOCK_SIZE];
    /* *
     * shared_image logicamente divisa come un blocco 8x8 su una sola riga
     * Sono 8 blocchi 8x8, 8 righe da 64 float.
     * [[image_block_0]
     *  [image_block_i]
     *  [image_block_7]]
     * */
    // CUDA related vars (ids)
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;
    // Image size related vars ()
    int imgDimX = img_size / blockDim.x; // numero di blocchi 8x8 sull asse X
    int imgDimY = img_size / blockDim.y; // numero di blocchi 8x8 sull'asse Y
    int imageIdY = global / BLOCK_SIZE / imgDimX; // indicizzazione del blocco, Y
    int imageIdX = global / BLOCK_SIZE % imgDimY; // indicizzazione del blocco, X
    int offset_y = imageIdY * BLOCK_SIZE * img_size; // si "sposta" verso il basso di (BLOCK_SIZE * img_size)
    int offset_x = imageIdX * BLOCK_SIZE; // si "sposta" verso destra di BLOCK_SIZE

    float sums = 0;
    shared_transform[threadIdx.y * blockDim.x + threadIdx.x] = transform_matrix[threadIdx.y * blockDim.x + threadIdx.x];
    for(int i = 0;i < BLOCK_SIZE;i++)
        shared_image[threadIdx.y*(BLOCK_SIZE*BLOCK_SIZE) + (threadIdx.x*BLOCK_SIZE) + i] = image_matrix[(offset_y+offset_x) + (threadIdx.x*img_size) + i];
        //shared_image[VA CAMBIATO] = image_matrix[(offset_y) + (threadIdx.y*img_size) + (threadIdx.x*BLOCK_SIZE) + i];
    __syncthreads();
    /* *
     * Il seguente IF serve per evitare accessi illegali alla memoria.
     * Essendo che adesso i Threads sono mappati sul blocco immagine,
     * potrebbero esserci thread che non svolgono lavoro.
     * In tal caso bisogna evitare che accedano alla memoria.
     * */
    if (global >= imgDimX * imgDimY * BLOCK_SIZE)return;


    // COLONNE DI T PER COLONNE DI IMG (T.T @ X)
    for (int i = 0;i < BLOCK_SIZE;i++) {
        for (int j = 0;j < BLOCK_SIZE;j++) {
            // sums += TX [ sempre la stessa colonna ] * T [ colonne in sequenza ]
            //sums += shared_transform[threadIdx.x + j * BLOCK_SIZE] * image_matrix[i + (offset_y + offset_x) + (j * img_size)];
            sums += shared_transform[threadIdx.x + j * BLOCK_SIZE] * shared_image[threadIdx.y*(BLOCK_SIZE*BLOCK_SIZE) + i + j*BLOCK_SIZE];
        }
        // TX [ riga ] = T[ colonna ] * X[ colonne ] (TX[riga] = somma dei prodotti)
        // shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + i] = sums;
        riga[i]=sums;
        sums = 0;
    }

    sums = 0;

    // RIGHE DI TX PER COLONNE DI T (TX @ T)
    for (int i = 0;i < BLOCK_SIZE;i++) {
        for (int j = 0;j < BLOCK_SIZE;j++) {
            // sums += TX [ sempre la stessa riga ] * T [ colonne in sequenza ]
            // sums += shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + j] * transform_matrix[i + j * BLOCK_SIZE];
            sums += riga[j] * shared_transform[i + j * 8];
        }
        // result [ riga ] = TX [ riga ] * T [ colonne ]
        result[(offset_y + offset_x) + (threadIdx.x * img_size) + i] = sums;
        sums = 0;
    }
}

void dct_all_blocks_cuda(float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result)
{
    // Pre-alloca memoria GPU per i blocchi temporanei
    float* temp2;
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_height / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(gridx, gridy);
    int mono_grid_Dim = ((gridx * gridy * 8)+(BLOCK_SIZE*BLOCK_SIZE)-1)/(BLOCK_SIZE*BLOCK_SIZE);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // subsampling (--128)
    sub_matrix_scalar<<<gridDim, blockDim>>>(image_matrix, 128, image_matrix, img_width * img_height);

    // applica la DCT
    cuda_matrix_dct_paper<<<mono_grid_Dim, blockDim>>>(image_matrix, img_width, transform_matrix, temp2);

    // Applicazione della quantizzazione
    float q_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};

    // alloca quant_matrix on device
    float* d_Q_matrix;
    CHECK_CUDA(hipMalloc(&d_Q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Q_matrix, q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Lancio del kernel quantizzazione
    divide_matrices<<<gridDim, blockDim>>>(temp2, d_Q_matrix, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("DCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp2));
    CHECK_CUDA(hipFree(d_Q_matrix));
}

void idct_all_blocks_cuda(const float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result)
{
    // Pre-alloca memoria GPU per i blocchi temporanei
    float* temp2;
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));

    // Applicazione della de-quantizzazione
    float q_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99 };

    // alloca quant_matrix on device
    float* d_Q_matrix;
    CHECK_CUDA(hipMalloc(&d_Q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Q_matrix, q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_height / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(gridx, gridy);
    int mono_grid_Dim = ((gridx * gridy * 8)+(BLOCK_SIZE*BLOCK_SIZE)-1)/(BLOCK_SIZE*BLOCK_SIZE);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // Lancio del kernel de-quantizzazione
    multiply_matrices<<<gridDim, blockDim>>>(image_matrix, d_Q_matrix, temp2, img_width * img_height);

    // applica la IDCT
    cuda_matrix_idct_paper<<<mono_grid_Dim, blockDim>>>(temp2, img_width, transform_matrix, result);

    // inverse of subsampling (++128)
    add_matrix_scalar<<<gridDim, blockDim>>>(result, 128, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("IDCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp2));
    CHECK_CUDA(hipFree(d_Q_matrix));
}