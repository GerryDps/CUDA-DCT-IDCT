#include "hip/hip_runtime.h"
//%%cuda --compiler-args "--library cublas --library jpeg -arch=sm_75"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <jpeglib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Check CUDA error
#define CHECK_CUDA(call)                                          \
    {                                                             \
        hipError_t err = call;                                   \
        if (err != hipSuccess)                                   \
        {                                                         \
            printf("%s : %d", hipGetErrorString(err), __LINE__); \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    }

// Check CUBLAS error
#define CHECK_CUBLAS(call)                                         \
    {                                                              \
        hipblasStatus_t err = call;                                 \
        if (err != HIPBLAS_STATUS_SUCCESS)                          \
        {                                                          \
            printf("CUBLAS error in %s : %d", __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    }

#define BLOCK_SIZE 8
#define IMAGE_SIZE 256

__constant__ float const_quant_matrix[BLOCK_SIZE*BLOCK_SIZE];

// Kernels CUDA per le operazioni aritmetiche element-wise
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, int size);
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, int size);

__global__ void divide_matrices(const float* A, const float* B, float* C, int size);
__global__ void multiply_matrices(const float* A, const float* B, float* C, int size);

// Host function to load image as matrix
static unsigned char *load_jpeg_as_matrix(const char *filename, int *width, int *height, int *channels);

// Host function to save matrix as jpeg
int save_grayscale_jpeg(const char *filename, unsigned char *image_matrix, int width, int height, int quality);

// Utils
void convertToFloat(unsigned char *input, float *output, int size);
void convertToUnsignedChar(const float *image_float, unsigned char *image_char, int size);

// Using CUBLAS HANDLE to compute the DCT and the IDCT
__host__ void dct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle);
__host__ void idct_all_blocks(const float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle);

int main()
{

    const char *filename = "camera256.tif.jpeg";
    int width, height, channels;

    /*// Load a jpeg image in image_matrix
    unsigned char *image_matrix = load_jpeg_as_matrix(filename, &width, &height, &channels);
    if (!image_matrix)
    {
        exit(EXIT_FAILURE);
    }

    // allocate host memory for the float image
    float *image_matrix_float;
    image_matrix_float = (float *)malloc(width * height * sizeof(float));
    convertToFloat(image_matrix, image_matrix_float, width * height * channels);
    free(image_matrix);*/

    width = 4096;
    height = 4096;

    float* image_matrix_float;
    image_matrix_float = (float*)malloc(width * height * sizeof(float));
    srand(41);
    for (int i = 0; i < height;i++) {
        for (int j = 0; j < width; j++) {
            image_matrix_float[i * width + j] = rand() % 256;;
        }
    }

    printf("Printing the 8x8 of image[] (matrix from the jpeg image w:%d h:%d)\n",width,height);
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", image_matrix_float[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Quantization matrix (su constant)
    float quant_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(const_quant_matrix), quant_matrix, sizeof(quant_matrix)));

    // Transform matrix (hardcoded for simplicity)
    float transform_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339,
            0.5, 0.5, 0, 0, 0, 0, -0.5, -0.5,
            0.4472136, 0.2236068, -0.2236068, -0.4472136, -0.4472136, -0.2236068, 0.2236068, 0.4472136,
            0, 0, -0.70710678, 0, 0, 0.70710678, 0, 0,
            0.35355339, -0.35355339, -0.35355339, 0.35355339, 0.35355339, -0.35355339, -0.35355339, 0.35355339,
            0.5, -0.5, 0, 0, 0, 0, 0.5, -0.5,
            0.2236068, -0.4472136, 0.4472136, -0.2236068, -0.2236068, 0.4472136, -0.4472136, 0.2236068,
            0, 0, 0, -0.70710678, 0.70710678, 0, 0, 0};

    // Initialize CUBLAS
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // allocate host memory for the result image
    float *result;
    result = (float *)malloc(width * height * sizeof(float));

    // allocate device memory for: image_block (d_A), trasform_matrix (d_B), and result (d_C)
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, width * height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, width * height * sizeof(float)));

    // copy host memory to device
    CHECK_CUDA(hipMemcpy(d_A, image_matrix_float, width * height * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, transform_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Compute DCT using CUBLAS
    // d_A = image_block | d_B = transform_matrix | d_C = result
    dct_all_blocks(d_A, height,width,d_B, d_C, handle);

    // copy result form device memory back to host
    // result = d_C
    CHECK_CUDA(hipMemcpy(result, d_C, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the dct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // d_E = result of the idct applied on the block_image compressed
    float *d_E;
    CHECK_CUDA(hipMalloc(&d_E, width * height * sizeof(float)));

    // Compute the idct
    // d_C = result ; d_B = transform_matrix ; d_E = result
    idct_all_blocks(d_C, height, width, d_B,d_E,handle);

    // copy device memory to host
    // result = d_E
    CHECK_CUDA(hipMemcpy(result, d_E, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the idct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Salva l'immagine in formato JPEG
    const char *filename_out = "output.jpg";
    int quality = 100; // Qualità JPEG (0-100)

    // allocate host memory for the usigned char image
    unsigned char *image_matrix_uc;
    image_matrix_uc = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    convertToUnsignedChar(result, image_matrix_uc, width * height);
    free(result);

    printf("Printing the 8x8 of U_C[] (unsignedchar)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%d ", image_matrix_uc[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    if (save_grayscale_jpeg(filename_out, image_matrix_uc, width, height, quality))
    {
        printf("Image saved successfully to %s\n", filename_out);
    }
    else
    {
        fprintf(stderr, "Error: Failed to save image\n");
    }


    // Cleanup
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_E));
    free(image_matrix_uc);
    free(image_matrix_float);
    return 0;
}

// Host function to load image as matrix
static unsigned char *load_jpeg_as_matrix(const char *filename, int *width, int *height, int *channels)
{
    struct jpeg_decompress_struct cinfo;
    struct jpeg_error_mgr jerr;

    FILE *infile = fopen(filename, "rb");
    if (!infile)
    {
        fprintf(stderr, "Error: Unable to open file %s\n", filename);
        return NULL;
    }

    // Set up error handling
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_decompress(&cinfo);

    // Specify the source of the data (the input file)
    jpeg_stdio_src(&cinfo, infile);

    // Read the JPEG header to get image info
    jpeg_read_header(&cinfo, TRUE);

    // Start decompression
    jpeg_start_decompress(&cinfo);

    *width = cinfo.output_width;
    *height = cinfo.output_height;
    *channels = cinfo.output_components; // 1 for grayscale, 3 for RGB
    int colorspace = cinfo.out_color_space;

    // Allocate memory for the pixel matrix
    unsigned long matrix_size = (*width) * (*height) * (*channels);
    unsigned char *image_matrix = (unsigned char *)malloc(matrix_size);
    if (!image_matrix)
    {
        fprintf(stderr, "Error: Unable to allocate memory for image matrix\n");
        jpeg_destroy_decompress(&cinfo);
        fclose(infile);
        return NULL;
    }

    // Read scanlines into the matrix row by row
    unsigned char *row_pointer[1];
    while (cinfo.output_scanline < cinfo.output_height)
    {
        row_pointer[0] = image_matrix + (cinfo.output_scanline * (*width) * (*channels));
        jpeg_read_scanlines(&cinfo, row_pointer, 1);
    }

    // Finish decompression and clean up
    jpeg_finish_decompress(&cinfo);
    jpeg_destroy_decompress(&cinfo);
    fclose(infile);

    return image_matrix;
}

int save_grayscale_jpeg(const char *filename, unsigned char *image_matrix, int width, int height, int quality)
{
    // Strutture di compressione JPEG
    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;

    // Imposta il gestore degli errori
    cinfo.err = jpeg_std_error(&jerr);

    // Inizializza l'oggetto di compressione JPEG
    jpeg_create_compress(&cinfo);

    // Apri il file in scrittura
    FILE *outfile = fopen(filename, "wb");
    if (!outfile)
    {
        fprintf(stderr, "Error: Unable to open file %s for writing\n", filename);
        return 0;
    }

    // Associa il file di output al compressore
    jpeg_stdio_dest(&cinfo, outfile);

    // Imposta i parametri dell'immagine JPEG
    cinfo.image_width = width;            // Larghezza in pixel
    cinfo.image_height = height;          // Altezza in pixel
    cinfo.input_components = 1;           // Numero di canali (1 per scala di grigi)
    cinfo.in_color_space = JCS_GRAYSCALE; // Colore: scala di grigi

    // Imposta i parametri di default e modifica la qualità
    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, quality, TRUE);

    // Inizia la compressione
    jpeg_start_compress(&cinfo, TRUE);

    // Scrive ogni riga dell'immagine
    while (cinfo.next_scanline < cinfo.image_height)
    {
        unsigned char *row_pointer = &image_matrix[cinfo.next_scanline * width];
        jpeg_write_scanlines(&cinfo, &row_pointer, 1);
    }

    // Termina la compressione
    jpeg_finish_compress(&cinfo);

    // Libera le risorse
    fclose(outfile);
    jpeg_destroy_compress(&cinfo);

    return 1;
}

// Convert an unsigned char image matrix to an float image matrix
void convertToFloat(unsigned char *input, float *output, int size)
{
    for (int i = 0; i < size; i++) {
        output[i] = (float)input[i];
    }
}

// Convert a float image matrix to unsigned char image matrix
void convertToUnsignedChar(const float *image_float, unsigned char *image_char, int size) {
    for (int i = 0; i < size; i++) {
        image_char[i] = (unsigned char)fminf(fmaxf(image_float[i], 0.0f), 255.0f); // Clamp tra 0 e 255
        //image_char[i] = (unsigned char)image_float[i]; // Clamp tra 0 e 255
    }
}

// Kernel CUDA per la sottrazione element-wise matrice - scalare
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] - scalar;
    }
}

// Kernel CUDA per l'addizione element-wise matrice - scalare
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] + scalar;
    }
}

// Kernel CUDA per la divisione elemento per elemento
__global__ void divide_matrices(const float* A, const float* B, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  round(A[global] / B[threadIdx.x * blockDim.x + threadIdx.y]);
    }
}

// Kernel CUDA per la moltiplicazione elemento per elemento
__global__ void multiply_matrices(const float* A, const float* B, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] * B[threadIdx.x * blockDim.x + threadIdx.y];
    }
}

/* *
 * Funzione per l'applicazione della "compressione" (IN_IMAGE->scaling->dct->quantization->OUT_COMPRESSED)
 * L'utilizzo di una matrice di quantizzazione con tutti 1 ad esempio, ovviamente non porta a nessuna compressione
 * e/o perdita di qualità, in quando dct/idct sono reversibili a meno di piccoli errori di arrotondamento.
 * In questo caso particolare la matrice di quantizzazione è quella definita da JPEG (diversa da 1),
 * e quindi introduce compressione/perdita di qualità.
 * */
void dct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    // Pre-alloca memoria GPU per i blocchi temporanei
    float *temp1, *temp2,*d_Q_matrix;
    CHECK_CUDA(hipMalloc(&temp1, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipGetSymbolAddress((void**)&d_Q_matrix,const_quant_matrix));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE);
    dim3 gridDim(gridx,gridy);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // subsampling (--128)
    sub_matrix_scalar<<<gridDim,blockDim>>>(image_matrix, 128, image_matrix, img_width * img_height);

    // Itera sui blocchi 8x8 - applica la DCT
    for (int block_row = 0; block_row < img_height; block_row += BLOCK_SIZE)
    {
        for (int block_col = 0; block_col < img_width; block_col += BLOCK_SIZE)
        {
            // Calcola l'offset del blocco corrente
            const float *image_block = image_matrix + block_row * img_width + block_col;
            float *result_block = temp2 + block_row * img_width + block_col;

            // Calcola temp1 = transform_matrix @ image_block
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, transform_matrix, BLOCK_SIZE, image_block, img_width,
                                     &beta, temp1, BLOCK_SIZE));

            // Calcola temp1 @ transform_matrix.T
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, temp1, BLOCK_SIZE, transform_matrix, BLOCK_SIZE,
                                     &beta, result_block, img_width));
        }
    }

    // Lancio del kernel quantizzazione
    divide_matrices<<<gridDim,blockDim>>>(temp2, d_Q_matrix, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("DCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);


    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp1));
    CHECK_CUDA(hipFree(temp2));
}

/* *
 * Funzione per l'applicazione della "de-compressione" (IN_COMPRESSED->de-quantization->idct->UPscaling->OUT_IMAGE)
 * */
void idct_all_blocks(const float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    // Pre-alloca memoria GPU per i blocchi temporanei
    float *temp1, *temp2,*d_Q_matrix;;
    CHECK_CUDA(hipMalloc(&temp1, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipGetSymbolAddress((void**)&d_Q_matrix,const_quant_matrix));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE);
    dim3 gridDim(gridx,gridy);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // Lancio del kernel de-quantizzazione
    multiply_matrices<<<gridDim,blockDim>>>(image_matrix, d_Q_matrix, temp2, img_width * img_height);

    // Itera sui blocchi 8x8 - applica la IDCT
    for (int block_row = 0; block_row < img_height; block_row += BLOCK_SIZE)
    {
        for (int block_col = 0; block_col < img_width; block_col += BLOCK_SIZE)
        {
            // Calcola l'offset del blocco corrente
            const float *image_block = temp2 + block_row * img_width + block_col;
            float *result_block = result + block_row * img_width + block_col;

            // Compute temp1 = transform_matrix.T @ image_block
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, transform_matrix, BLOCK_SIZE, image_block, img_width,
                                     &beta, temp1, BLOCK_SIZE));

            // Compute temp1 @ transform_matrix
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, temp1, BLOCK_SIZE, transform_matrix, BLOCK_SIZE,
                                     &beta, result_block, img_width));
        }
    }

    // inverse of subsampling (++128)
    add_matrix_scalar<<<gridDim,blockDim>>>(result, 128, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("IDCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp1));
    CHECK_CUDA(hipFree(temp2));
}
