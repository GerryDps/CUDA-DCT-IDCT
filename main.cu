#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <jpeglib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

// Check CUDA error
#define CHECK_CUDA(call)                                          \
    {                                                             \
        hipError_t err = call;                                   \
        if (err != hipSuccess)                                   \
        {                                                         \
            printf("%s : %d", hipGetErrorString(err), __LINE__); \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    }

// Check CUBLAS error
#define CHECK_CUBLAS(call)                                         \
    {                                                              \
        hipblasStatus_t err = call;                                 \
        if (err != HIPBLAS_STATUS_SUCCESS)                          \
        {                                                          \
            printf("CUBLAS error in %s : %d", __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    }

#define BLOCK_SIZE 8
#define IMAGE_SIZE 256

// Kernels CUDA per le operazioni aritmetiche element-wise
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, int size);
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, int size);

__global__ void divide_matrices(const float* A, const float* B, float* C, int size);
__global__ void multiply_matrices(const float* A, const float* B, float* C, int size);

// Host function to load image as matrix
static unsigned char *load_jpeg_as_matrix(const char *filename, int *width, int *height, int *channels);

// Host function to save matrix as jpeg
int save_grayscale_jpeg(const char *filename, unsigned char *image_matrix, int width, int height, int quality);

// Utils
void convertToFloat(unsigned char *input, float *output, int size);
void convertToUnsignedChar(const float *image_float, unsigned char *image_char, int size);

// Using CUBLAS HANDLE to compute the DCT and the IDCT
void dct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle);
void idct_all_blocks(const float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle);

// Kernels for dct / idct
__global__ void cuda_matrix_dct(const float* image_matrix, const float* transform_matrix, float* result);
__global__ void cuda_matrix_idct(const float* image_matrix, const float* transform_matrix, float* result);

// Using cuda kernels to compute the DCT and the IDCT
void dct_all_blocks_cuda(float* image_matrix, int img_height, int img_width, const float* transform_matrix, float* result);
void idct_all_blocks_cuda(const float* image_matrix, int img_height, int img_width, const float* transform_matrix, float* result);

int main()
{

    const char *filename = "camera256.tif.jpeg";
    int width, height, channels;

    // Load a jpeg image in image_matrix
    unsigned char *image_matrix = load_jpeg_as_matrix(filename, &width, &height, &channels);
    if (!image_matrix)
    {
        exit(EXIT_FAILURE);
    }

    // allocate host memory for the float image
    float *image_matrix_float;
    image_matrix_float = (float *)malloc(width * height * sizeof(float));
    convertToFloat(image_matrix, image_matrix_float, width * height * channels);
    free(image_matrix);

    printf("Printing the 8x8 of image[] (matrix coming from the jpeg image)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", image_matrix_float[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    /* image_matrix
    float image_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            156, 159, 158, 155, 158, 156, 159, 158,
            160, 154, 157, 158, 157, 159, 158, 158,
            156, 159, 157, 155, 157, 157, 160, 158,
            160, 154, 157, 158, 157, 160, 158, 158,
            157, 152, 155, 158, 159, 155, 156, 155,
            155, 155, 155, 157, 156, 159, 152, 158,
            156, 154, 157, 156, 153, 155, 154, 155,
            159, 159, 156, 158, 156, 159, 157, 160};
    */

    // Quantization matrix (fare __device__ )
    float quant_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};

    // Transform matrix (hardcoded for simplicity)
    float transform_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339,
            0.5, 0.5, 0, 0, 0, 0, -0.5, -0.5,
            0.4472136, 0.2236068, -0.2236068, -0.4472136, -0.4472136, -0.2236068, 0.2236068, 0.4472136,
            0, 0, -0.70710678, 0, 0, 0.70710678, 0, 0,
            0.35355339, -0.35355339, -0.35355339, 0.35355339, 0.35355339, -0.35355339, -0.35355339, 0.35355339,
            0.5, -0.5, 0, 0, 0, 0, 0.5, -0.5,
            0.2236068, -0.4472136, 0.4472136, -0.2236068, -0.2236068, 0.4472136, -0.4472136, 0.2236068,
            0, 0, 0, -0.70710678, 0.70710678, 0, 0, 0};

    // Initialize CUBLAS
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // allocate host memory for the result image
    float *result;
    result = (float *)malloc(width * height * sizeof(float));

    // allocate device memory for: image_block (d_A), trasform_matrix (d_B), and result (d_C)
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, width * height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, width * height * sizeof(float)));

    // copy host memory to device
    CHECK_CUDA(hipMemcpy(d_A, image_matrix_float, width * height * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, transform_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Compute DCT using CUBLAS
    // d_A = image_block ; d_B = transform_matrix ; d_C = result
    //dct_all_blocks(d_A, height,width,d_B, d_C, handle);
    dct_all_blocks_cuda(d_A, height,width,d_B, d_C);

    // copy device memory to host
    // result = d_C
    CHECK_CUDA(hipMemcpy(result, d_C, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the dct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // d_E = result of the idct applied on the block_image compressed
    float *d_E;
    CHECK_CUDA(hipMalloc(&d_E, width * height * sizeof(float)));

    // Compute the idct
    // d_C = result ; d_B = transform_matrix ; d_E = result
    //idct_all_blocks(d_C, height, width, d_B,d_E,handle);
    idct_all_blocks_cuda(d_C, height, width, d_B,d_E);

    // copy device memory to host
    // result = d_E
    CHECK_CUDA(hipMemcpy(result, d_E, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the idct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Salva l'immagine in formato JPEG
    const char *filename_out = "output.jpg";
    int quality = 100; // Qualità JPEG (0-100)

    // allocate host memory for the usigned char image
    unsigned char *image_matrix_uc;
    image_matrix_uc = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    convertToUnsignedChar(result, image_matrix_uc, width * height);
    free(result);

    printf("Printing the 8x8 of U_C[] (unsignedchar)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%d ", image_matrix_uc[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    if (save_grayscale_jpeg(filename_out, image_matrix_uc, width, height, quality))
    {
        printf("Image saved successfully to %s\n", filename_out);
    }
    else
    {
        fprintf(stderr, "Error: Failed to save image\n");
    }


    // Cleanup
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}

// Host function to load image as matrix
static unsigned char *load_jpeg_as_matrix(const char *filename, int *width, int *height, int *channels)
{
    struct jpeg_decompress_struct cinfo;
    struct jpeg_error_mgr jerr;

    FILE *infile = fopen(filename, "rb");
    if (!infile)
    {
        fprintf(stderr, "Error: Unable to open file %s\n", filename);
        return NULL;
    }

    // Set up error handling
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_decompress(&cinfo);

    // Specify the source of the data (the input file)
    jpeg_stdio_src(&cinfo, infile);

    // Read the JPEG header to get image info
    jpeg_read_header(&cinfo, TRUE);

    // Start decompression
    jpeg_start_decompress(&cinfo);

    *width = cinfo.output_width;
    *height = cinfo.output_height;
    *channels = cinfo.output_components; // 1 for grayscale, 3 for RGB
    int colorspace = cinfo.out_color_space;
    printf("Color Space: %d\n", colorspace);

    // Allocate memory for the pixel matrix
    unsigned long matrix_size = (*width) * (*height) * (*channels);
    unsigned char *image_matrix = (unsigned char *)malloc(matrix_size);
    if (!image_matrix)
    {
        fprintf(stderr, "Error: Unable to allocate memory for image matrix\n");
        jpeg_destroy_decompress(&cinfo);
        fclose(infile);
        return NULL;
    }

    // Read scanlines into the matrix row by row
    unsigned char *row_pointer[1];
    while (cinfo.output_scanline < cinfo.output_height)
    {
        row_pointer[0] = image_matrix + (cinfo.output_scanline * (*width) * (*channels));
        jpeg_read_scanlines(&cinfo, row_pointer, 1);
    }

    // Finish decompression and clean up
    jpeg_finish_decompress(&cinfo);
    jpeg_destroy_decompress(&cinfo);
    fclose(infile);

    return image_matrix;
}

int save_grayscale_jpeg(const char *filename, unsigned char *image_matrix, int width, int height, int quality)
{
    // Strutture di compressione JPEG
    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;

    // Imposta il gestore degli errori
    cinfo.err = jpeg_std_error(&jerr);

    // Inizializza l'oggetto di compressione JPEG
    jpeg_create_compress(&cinfo);

    // Apri il file in scrittura
    FILE *outfile = fopen(filename, "wb");
    if (!outfile)
    {
        fprintf(stderr, "Error: Unable to open file %s for writing\n", filename);
        return 0;
    }

    // Associa il file di output al compressore
    jpeg_stdio_dest(&cinfo, outfile);

    // Imposta i parametri dell'immagine JPEG
    cinfo.image_width = width;            // Larghezza in pixel
    cinfo.image_height = height;          // Altezza in pixel
    cinfo.input_components = 1;           // Numero di canali (1 per scala di grigi)
    cinfo.in_color_space = JCS_GRAYSCALE; // Colore: scala di grigi

    // Imposta i parametri di default e modifica la qualità
    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, quality, TRUE);

    // Inizia la compressione
    jpeg_start_compress(&cinfo, TRUE);

    // Scrive ogni riga dell'immagine
    while (cinfo.next_scanline < cinfo.image_height)
    {
        unsigned char *row_pointer = &image_matrix[cinfo.next_scanline * width];
        jpeg_write_scanlines(&cinfo, &row_pointer, 1);
    }

    // Termina la compressione
    jpeg_finish_compress(&cinfo);

    // Libera le risorse
    fclose(outfile);
    jpeg_destroy_compress(&cinfo);

    return 1;
}

// Convert an unsigned char image matrix to an float image matrix
void convertToFloat(unsigned char *input, float *output, int size)
{
    for (int i = 0; i < size; i++) {
        output[i] = (float)input[i];
    }
}

// Convert a float image matrix to unsigned char image matrix
void convertToUnsignedChar(const float *image_float, unsigned char *image_char, int size) {
    for (int i = 0; i < size; i++) {
        image_char[i] = (unsigned char)fminf(fmaxf(image_float[i], 0.0f), 255.0f); // Clamp tra 0 e 255
        //image_char[i] = (unsigned char)image_float[i]; // Clamp tra 0 e 255
    }
}

/*quantization_matrix[8][8]
Id_x = gridId.x * blockDim.x + threadIdx.x
Id_y = gridID.y * blockDim.y + threadIdx.y
global = Id_y * gridDim.x * blockDim.x + Id_x
C[global] =  A[global] / quantization_matrix[threadIdx.x * BLOCK_SIZE + threadIdx.y]*/

// Kernel CUDA per la sottrazione element-wise matrice - scalare
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] - scalar;
    }
}

// Kernel CUDA per l'addizione element-wise matrice - scalare
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] + scalar;
    }
}

// Kernel CUDA per la divisione elemento per elemento
__global__ void divide_matrices(const float* A, const float* B, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  round(A[global] / B[threadIdx.x * blockDim.x + threadIdx.y]);
    }
}

// Kernel CUDA per la moltiplicazione elemento per elemento
__global__ void multiply_matrices(const float* A, const float* B, float* C, int size) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] * B[threadIdx.x * blockDim.x + threadIdx.y];
    }
}

/* *
 * Effettua la DCT utilizzando la matrice di trasformazione
 * (TRASFORM_MATRIX @ IMAGE_MATRIX) @ TRANSFORM_MATRIX.T
 * La matrice di trasformazione è 8x8
 * result = TRASFORM_MATRIX @ IMAGE_MATRIX
 * result = result @ TRANSFORM_MATRIX.T
 * */
__global__ void cuda_matrix_dct(const float* image_matrix, const float* transform_matrix, float* result) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;
    int offset_x = blockIdx.x * blockDim.x;
    int offset_y = blockIdx.y * blockDim.y;

    float sums = 0;

    // result = transform_matrix @ image_matrix
    // result = transform_matrix[righe] @ image_matrix[colonne]
    for (int i = 0;i < blockDim.x;i++) {
        sums += transform_matrix[threadIdx.y * blockDim.x + i] * image_matrix[(offset_y * gridDim.x * blockDim.x) + i * (gridDim.x * blockDim.x) + Id_x];
    }
    result[Id_y * gridDim.x * blockDim.x + Id_x] = sums;
    sums = 0;
    // Devo attendere il completamento della DOT precedente
    __syncthreads();

    // result = result(precedente) @ transform_matrix.T (trasposta)
    // result = result(precedente)[righe] @ transform_matrix[righe] (perchè la trasposta)
    // result shared per fare 8 letture shared
    for (int i = 0;i < blockDim.x;i++) {
        sums += result[Id_y * (gridDim.x * blockDim.x) + offset_x + i] * transform_matrix[threadIdx.x * blockDim.x + i];
    }
    // Non possono sovrascrivere prima che abbiano finito tutto altrimenti leggerebbero una riga sbagliata
    __syncthreads();
    result[Id_y * gridDim.x * blockDim.x + Id_x] = sums;
    //result_g[Id_y * gridDim.x * blockDim.x + Id_x] = sums; implementare result_shared
}

/* *
 * Effettua la IDCT utilizzando la matrice di trasformazione
 * (TRANSFORM_MATRIX.T @ DCT_MATRIX) @ TRANSFORM_MATRIX
 * La matrice di trasformazione è 8x8
 * result = TRANSFORM_MATRIX.T @ DCT_MATRIX
 * result = result @ TRANSFORM_MATRIX
 * */
__global__ void cuda_matrix_idct(const float* image_matrix, const float* transform_matrix, float* result) {
    // Calcola l'indice globale del thread
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;
    int offset_x = blockIdx.x * blockDim.x;
    int offset_y = blockIdx.y * blockDim.y;

    float sums = 0;

    // result = transform_matrix.T @ dct_matrix
    // result = transform_matrix[colonne](x trasposta) @ image_matrix[colonne]
    for (int i = 0;i < blockDim.x;i++) {
        sums += transform_matrix[i * blockDim.x + threadIdx.y] * image_matrix[(offset_y * gridDim.x * blockDim.x) + i * (gridDim.x * blockDim.x) + Id_x];
    }
    result[Id_y * gridDim.x * blockDim.x + Id_x] = sums;
    sums = 0;
    __syncthreads();

    // result = result(precedente) @ transform_matrix
    // result = result[righe] @ transform_matrix[colonne]
    for (int i = 0;i < blockDim.x;i++) {
        sums += result[Id_y * (gridDim.x * blockDim.x) + offset_x + i] * transform_matrix[i * blockDim.x + threadIdx.x];
    }
    // Non possono sovrascrivere prima che abbiano finito tutto altrimenti leggerebbero una riga sbagliata
    __syncthreads();
    result[Id_y * gridDim.x * blockDim.x + Id_x] = sums;
    //result_global[Id_y * gridDim.x * blockDim.x + Id_x] = sums; implementare result_shared
}

void dct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    // Pre-alloca memoria GPU per i blocchi temporanei
    float *temp1, *temp2;
    CHECK_CUDA(hipMalloc(&temp1, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE);
    dim3 gridDim(gridx,gridy);

    // subsampling (--128)
    sub_matrix_scalar<<<gridDim,blockDim>>>(image_matrix, 128, image_matrix, img_width * img_height);

    // Itera sui blocchi 8x8 - applica la DCT
    for (int block_row = 0; block_row < img_height; block_row += BLOCK_SIZE)
    {
        for (int block_col = 0; block_col < img_width; block_col += BLOCK_SIZE)
        {
            // Calcola l'offset del blocco corrente
            const float *image_block = image_matrix + block_row * img_width + block_col;
            float *result_block = temp2 + block_row * img_width + block_col;

            // Calcola temp1 = transform_matrix @ image_block
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, transform_matrix, BLOCK_SIZE, image_block, img_width,
                                     &beta, temp1, BLOCK_SIZE));

            // Calcola temp1 @ transform_matrix.T
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, temp1, BLOCK_SIZE, transform_matrix, BLOCK_SIZE,
                                     &beta, result_block, img_width));
        }
    }

    // Applicazione della quantizzazione
    float q_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};

    // alloca quant_matrix on device
    float *d_Q_matrix;
    CHECK_CUDA(hipMalloc(&d_Q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Q_matrix, q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Lancio del kernel quantizzazione
    divide_matrices<<<gridDim,blockDim>>>(temp2, d_Q_matrix, result, img_width * img_height);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp1));
    CHECK_CUDA(hipFree(temp2));
    CHECK_CUDA(hipFree(d_Q_matrix));
}

void idct_all_blocks(const float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    // Pre-alloca memoria GPU per i blocchi temporanei
    float *temp1, *temp2;
    CHECK_CUDA(hipMalloc(&temp1, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));

    // Applicazione della de-quantizzazione
    float q_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};

    // alloca quant_matrix on device
    float *d_Q_matrix;
    CHECK_CUDA(hipMalloc(&d_Q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Q_matrix, q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE);
    dim3 gridDim(gridx,gridy);

    // Lancio del kernel de-quantizzazione
    multiply_matrices<<<gridDim,blockDim>>>(image_matrix, d_Q_matrix, temp2, img_width * img_height);

    // Itera sui blocchi 8x8 - applica la IDCT
    for (int block_row = 0; block_row < img_height; block_row += BLOCK_SIZE)
    {
        for (int block_col = 0; block_col < img_width; block_col += BLOCK_SIZE)
        {
            // Calcola l'offset del blocco corrente
            const float *image_block = temp2 + block_row * img_width + block_col;
            float *result_block = result + block_row * img_width + block_col;

            // Compute temp1 = transform_matrix.T @ image_block
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, transform_matrix, BLOCK_SIZE, image_block, img_width,
                                     &beta, temp1, BLOCK_SIZE));

            // Compute temp1 @ transform_matrix
            CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE,
                                     &alpha, temp1, BLOCK_SIZE, transform_matrix, BLOCK_SIZE,
                                     &beta, result_block, img_width));


        }
    }

    // inverse of subsampling (++128)
    add_matrix_scalar<<<gridDim,blockDim>>>(result, 128, result, img_width * img_height);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp1));
    CHECK_CUDA(hipFree(temp2));
    CHECK_CUDA(hipFree(d_Q_matrix));
}

void dct_all_blocks_cuda(float* image_matrix, int img_height, int img_width, const float* transform_matrix, float* result)
{
    // Pre-alloca memoria GPU per i blocchi temporanei
    float* temp2;
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(gridx, gridy);

    // subsampling (--128)
    sub_matrix_scalar<<<gridDim, blockDim>>>(image_matrix, 128, image_matrix, img_width * img_height);

    // applica la DCT
    cuda_matrix_dct<<<gridDim, blockDim>>>(image_matrix, transform_matrix, temp2);

    // Applicazione della quantizzazione
    float q_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};

    // alloca quant_matrix on device
    float* d_Q_matrix;
    CHECK_CUDA(hipMalloc(&d_Q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Q_matrix, q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Lancio del kernel quantizzazione
    divide_matrices<<<gridDim, blockDim>>>(temp2, d_Q_matrix, result, img_width * img_height);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp2));
    CHECK_CUDA(hipFree(d_Q_matrix));
}

void idct_all_blocks_cuda(const float* image_matrix, int img_height, int img_width, const float* transform_matrix, float* result)
{
    // Pre-alloca memoria GPU per i blocchi temporanei
    float* temp2;
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));

    // Applicazione della de-quantizzazione
    float q_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99 };

    // alloca quant_matrix on device
    float* d_Q_matrix;
    CHECK_CUDA(hipMalloc(&d_Q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMemcpy(d_Q_matrix, q_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(gridx, gridy);

    // Lancio del kernel de-quantizzazione
    multiply_matrices<<<gridDim, blockDim>>>(image_matrix, d_Q_matrix, temp2, img_width * img_height);

    // applica la IDCT
    cuda_matrix_idct<<<gridDim, blockDim>>>(temp2, transform_matrix, result);

    // inverse of subsampling (++128)
    add_matrix_scalar<<<gridDim, blockDim>>>(result, 128, result, img_width * img_height);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp2));
    CHECK_CUDA(hipFree(d_Q_matrix));
}

/* *
 * Effettua la DCT utilizzando la matrice di trasformazione
 * (TRANSFORM_MATRIX @ IMAGE) @ TRANSFORM_MATRIX.T
 * La matrice di trasformazione è 8x8
 * shared_matrix = TRANSFORM_MATRIX @ IMAGE
 * result = shared_matrix @ TRANSFORM_MATRIX.T
 *
 * Questo kernel va chiamato passando la grandezza di __shared___:
 * cuda_matrix_idct_paper<<<gridDim, blockDim, width*height*sizeof(float)>>>
 * */
__global__ void cuda_matrix_dct_paper(const float* image_matrix, int img_size, const float* transform_matrix, float* result) {
    extern __shared__ float shared_matrix[];
    // CUDA related vars (ids)
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;
    // Image size related vars ()
    int imgDimX = img_size / blockDim.x; // numero di blocchi 8x8 sull asse X
    int imgDimY = img_size / blockDim.y; // numero di blocchi 8x8 sull'asse Y
    int imageIdY = global / 8 / imgDimX; // indicizzazione del blocco, Y
    int imageIdX = global / 8 % imgDimY; // indicizzazione del blocco, X
    int offset_y = imageIdY * BLOCK_SIZE * img_size; // si "sposta" verso il basso di (BLOCK_SIZE * img_size)
    int offset_x = imageIdX * BLOCK_SIZE; // si "sposta" verso destra di BLOCK_SIZE

    float sums = 0;
    /* *
     * Il seguente IF serve per evitare accessi illegali alla memoria.
     * Essendo che adesso i Threads sono mappati sul blocco immagine,
     * potrebbero esserci thread che non svolgono lavoro.
     * In tal caso bisogna evitare che accedano alla memoria.
     * */
    if (global >= imgDimX * imgDimY * 8)return;

    // RIGHE DI T PER COLONNE DI IMG (T @ X)
    for (int i = 0;i < 8;i++) {
        for (int j = 0;j < 8;j++) {
            // sums += T [ sempre la stessa riga ] * X [ colonne in sequenza ]
            sums += transform_matrix[threadIdx.x * 8 + j] * image_matrix[i + (offset_y + offset_x) + (j * img_size)];
        }
        // TX [ riga ] = T[ riga ] * X[ colonne ] (TX[riga] = somma dei prodotti)
        shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + i] = sums;
        sums = 0;
    }

    sums = 0;

    // RIGHE DI TX PER RIGHE DI T (TX @ T.T)
    for (int i = 0;i < 8;i++) {
        for (int j = 0;j < 8;j++) {
            // sums += TX [ sempre la stessa riga ] * T [ righe in sequenza ]
            sums += shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + j] * transform_matrix[i * 8 + j];
        }
        // result [ riga ] = TX [ riga ] * T [ righe ]
        result[(offset_y + offset_x) + (threadIdx.x * img_size) + i] = sums;
        sums = 0;
    }
}

/* *
 * Effettua la IDCT utilizzando la matrice di trasformazione
 * (TRANSFORM_MATRIX.T @ DCT_MATRIX) @ TRANSFORM_MATRIX
 * La matrice di trasformazione è 8x8
 * shared_matrix = TRANSFORM_MATRIX.T @ DCT_MATRIX
 * result = shared_matrix @ TRANSFORM_MATRIX
 *
 * Questo kernel va chiamato passando la grandezza di __shared___:
 * cuda_matrix_idct_paper<<<gridDim, blockDim, width*height*sizeof(float)>>>
 * */
__global__ void cuda_matrix_idct_paper(const float* image_matrix, int img_size,const float* transform_matrix, float* result) {
    extern __shared__ float shared_matrix[];
    // CUDA related vars (ids)
    int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    int global = Id_y * gridDim.x * blockDim.x + Id_x;
    // Image size related vars ()
    int imgDimX = img_size / blockDim.x; // numero di blocchi 8x8 sull asse X
    int imgDimY = img_size / blockDim.y; // numero di blocchi 8x8 sull'asse Y
    int imageIdY = global / 8 / imgDimX; // indicizzazione del blocco, Y
    int imageIdX = global / 8 % imgDimY; // indicizzazione del blocco, X
    int offset_y = imageIdY * BLOCK_SIZE * img_size; // si "sposta" verso il basso di (BLOCK_SIZE * img_size)
    int offset_x = imageIdX * BLOCK_SIZE; // si "sposta" verso destra di BLOCK_SIZE

    float sums = 0;
    /* *
     * Il seguente IF serve per evitare accessi illegali alla memoria.
     * Essendo che adesso i Threads sono mappati sul blocco immagine,
     * potrebbero esserci thread che non svolgono lavoro.
     * In tal caso bisogna evitare che accedano alla memoria.
     * */
    if (global >= imgDimX * imgDimY * 8)return;


    // COLONNE DI T PER COLONNE DI IMG (T.T @ X)
    for (int i = 0;i < 8;i++) {
        for (int j = 0;j < 8;j++) {
            // sums += TX [ sempre la stessa colonna ] * T [ colonne in sequenza ]
            sums += transform_matrix[threadIdx.x + j * 8] * image_matrix[i + (offset_y + offset_x) + (j * img_size)];
        }
        // TX [ riga ] = T[ colonna ] * X[ colonne ] (TX[riga] = somma dei prodotti)
        shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + i] = sums;
        sums = 0;
    }

    sums = 0;

    // RIGHE DI TX PER COLONNE DI T (TX @ T)
    for (int i = 0;i < 8;i++) {
        for (int j = 0;j < 8;j++) {
            // sums += TX [ sempre la stessa riga ] * T [ colonne in sequenza ]
            sums += shared_matrix[(offset_y + offset_x) + threadIdx.x * img_size + j] * transform_matrix[i + j * 8];
        }
        // result [ riga ] = TX [ riga ] * T [ colonne ]
        result[(offset_y + offset_x) + (threadIdx.x * img_size) + i] = sums;
        sums = 0;
    }
}