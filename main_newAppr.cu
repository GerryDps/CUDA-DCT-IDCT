#include "hip/hip_runtime.h"
//%%cuda --compiler-args "--library cublas --library jpeg -arch=sm_75"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <jpeglib.h>
#include <hip/hip_runtime.h>

// Check CUDA error
#define CHECK_CUDA(call)                                          \
    {                                                             \
        hipError_t err = call;                                   \
        if (err != hipSuccess)                                   \
        {                                                         \
            printf("%s : %d", hipGetErrorString(err), __LINE__); \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    }

#define BLOCK_SIZE 8
#define IMAGE_SIZE 256

__constant__ float const_quant_matrix[BLOCK_SIZE*BLOCK_SIZE];

// Kernels CUDA per le operazioni aritmetiche element-wise
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, const int size);
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, const int size);

__global__ void divide_matrices(const float* A, const float* B, float* C, const int size);
__global__ void multiply_matrices(const float* A, const float* B, float* C, const int size);

// Host function to load image as matrix
static unsigned char *load_jpeg_as_matrix(const char *filename, int *width, int *height, int *channels);

// Host function to save matrix as jpeg
int save_grayscale_jpeg(const char *filename, unsigned char *image_matrix, const int width, const int height, const int quality);

// Utils
void convertToFloat(const unsigned char *input, float *output, const int size);
void convertToUnsignedChar(const float *image_float, unsigned char *image_char, const int size);

// Using cuda kernels to compute the DCT and the IDCT
// Those FUNC use intenal dct/idct kernel function, differs according to implementation used.
void dct_all_blocks_cuda(float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result);
void idct_all_blocks_cuda(const float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result);

int main()
{
    const char *filename = "baboon.tif.jpeg";
    size_t width, height, channels;

    // Load a jpeg image in image_matrix
    unsigned char *image_matrix = load_jpeg_as_matrix(filename, &width, &height, &channels);
    if (!image_matrix)
    {
        exit(EXIT_FAILURE);
    }

    // allocate host memory for the float image
    float *image_matrix_float;
    image_matrix_float = (float *)malloc(width * height * sizeof(float));
    convertToFloat(image_matrix, image_matrix_float, width * height * channels);
    free(image_matrix);

    /*width = 4096;
    height = 4096;

    float* image_matrix_float;
    image_matrix_float = (float*)malloc(width * height * sizeof(float));
    srand(41);
    for (int i = 0; i < height;i++) {
        for (int j = 0; j < width; j++) {
            image_matrix_float[i * width + j] = rand() % 256;
        }
    }*/

    printf("Printing the 8x8 of image[] (matrix from the jpeg image w:%d h:%d)\n",width,height);
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", image_matrix_float[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Quantization matrix (su constant)
    float quant_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};
    // memoria costante
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(const_quant_matrix), quant_matrix, sizeof(quant_matrix)));

    // Transform matrix (hardcoded for simplicity)
    float transform_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339,
            0.5, 0.5, 0, 0, 0, 0, -0.5, -0.5,
            0.4472136, 0.2236068, -0.2236068, -0.4472136, -0.4472136, -0.2236068, 0.2236068, 0.4472136,
            0, 0, -0.70710678, 0, 0, 0.70710678, 0, 0,
            0.35355339, -0.35355339, -0.35355339, 0.35355339, 0.35355339, -0.35355339, -0.35355339, 0.35355339,
            0.5, -0.5, 0, 0, 0, 0, 0.5, -0.5,
            0.2236068, -0.4472136, 0.4472136, -0.2236068, -0.2236068, 0.4472136, -0.4472136, 0.2236068,
            0, 0, 0, -0.70710678, 0.70710678, 0, 0, 0};

    // allocate host memory for the result image
    float *result;
    result = (float *)malloc(width * height * sizeof(float));

    // allocate device memory for: image_block (d_A), trasform_matrix (d_B), and result (d_C)
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, width * height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, width * height * sizeof(float)));

    // copy host memory to device
    CHECK_CUDA(hipMemcpy(d_A, image_matrix_float, width * height * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, transform_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Compute DCT
    // d_A = image_block ; d_B = transform_matrix ; d_C = result
    dct_all_blocks_cuda(d_A, height,width,d_B, d_C);

    // copy device memory to host
    // result = d_C
    CHECK_CUDA(hipMemcpy(result, d_C, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the dct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // d_E = result of the idct applied on the block_image compressed
    float *d_E;
    CHECK_CUDA(hipMalloc(&d_E, width * height * sizeof(float)));

    // Compute the idct
    // d_C = result ; d_B = transform_matrix ; d_E = result
    idct_all_blocks_cuda(d_C, height, width, d_B,d_E);

    // copy device memory to host
    // result = d_E
    CHECK_CUDA(hipMemcpy(result, d_E, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the idct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Salva l'immagine in formato JPEG
    const char *filename_out = "output.jpg";
    int quality = 100; // Qualità JPEG (0-100)

    // allocate host memory for the usigned char image
    unsigned char *image_matrix_uc;
    image_matrix_uc = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    convertToUnsignedChar(result, image_matrix_uc, width * height);
    free(result);

    printf("Printing the 8x8 of U_C[] (unsignedchar)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%d ", image_matrix_uc[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    if (save_grayscale_jpeg(filename_out, image_matrix_uc, width, height, quality))
    {
        printf("Image saved successfully to %s\n", filename_out);
    }
    else
    {
        fprintf(stderr, "Error: Failed to save image\n");
    }
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_E));
    free(image_matrix_uc);
    free(image_matrix_float);
    return 0;
}

// Host function to load image as matrix
static unsigned char *load_jpeg_as_matrix(const char *filename, int *width, int *height, int *channels)
{
    struct jpeg_decompress_struct cinfo;
    struct jpeg_error_mgr jerr;

    FILE *infile = fopen(filename, "rb");
    if (!infile)
    {
        fprintf(stderr, "Error: Unable to open file %s\n", filename);
        return NULL;
    }

    // Set up error handling
    cinfo.err = jpeg_std_error(&jerr);
    jpeg_create_decompress(&cinfo);

    // Specify the source of the data (the input file)
    jpeg_stdio_src(&cinfo, infile);

    // Read the JPEG header to get image info
    jpeg_read_header(&cinfo, TRUE);

    // Start decompression
    jpeg_start_decompress(&cinfo);

    *width = cinfo.output_width;
    *height = cinfo.output_height;
    *channels = cinfo.output_components; // 1 for grayscale, 3 for RGB
    int colorspace = cinfo.out_color_space;
    printf("Color Space: %d\n", colorspace);

    // Allocate memory for the pixel matrix
    unsigned long matrix_size = (*width) * (*height) * (*channels);
    unsigned char *image_matrix = (unsigned char *)malloc(matrix_size);
    if (!image_matrix)
    {
        fprintf(stderr, "Error: Unable to allocate memory for image matrix\n");
        jpeg_destroy_decompress(&cinfo);
        fclose(infile);
        return NULL;
    }

    // Read scanlines into the matrix row by row
    unsigned char *row_pointer[1];
    while (cinfo.output_scanline < cinfo.output_height)
    {
        row_pointer[0] = image_matrix + (cinfo.output_scanline * (*width) * (*channels));
        jpeg_read_scanlines(&cinfo, row_pointer, 1);
    }

    // Finish decompression and clean up
    jpeg_finish_decompress(&cinfo);
    jpeg_destroy_decompress(&cinfo);
    fclose(infile);

    return image_matrix;
}

int save_grayscale_jpeg(const char *filename, unsigned char *image_matrix, const int width, const int height, const int quality)
{
    // Strutture di compressione JPEG
    struct jpeg_compress_struct cinfo;
    struct jpeg_error_mgr jerr;

    // Imposta il gestore degli errori
    cinfo.err = jpeg_std_error(&jerr);

    // Inizializza l'oggetto di compressione JPEG
    jpeg_create_compress(&cinfo);

    // Apri il file in scrittura
    FILE *outfile = fopen(filename, "wb");
    if (!outfile)
    {
        fprintf(stderr, "Error: Unable to open file %s for writing\n", filename);
        return 0;
    }

    // Associa il file di output al compressore
    jpeg_stdio_dest(&cinfo, outfile);

    // Imposta i parametri dell'immagine JPEG
    cinfo.image_width = width;            // Larghezza in pixel
    cinfo.image_height = height;          // Altezza in pixel
    cinfo.input_components = 1;           // Numero di canali (1 per scala di grigi)
    cinfo.in_color_space = JCS_GRAYSCALE; // Colore: scala di grigi

    // Imposta i parametri di default e modifica la qualità
    jpeg_set_defaults(&cinfo);
    jpeg_set_quality(&cinfo, quality, TRUE);

    // Inizia la compressione
    jpeg_start_compress(&cinfo, TRUE);

    // Scrive ogni riga dell'immagine
    while (cinfo.next_scanline < cinfo.image_height)
    {
        unsigned char *row_pointer = &image_matrix[cinfo.next_scanline * width];
        jpeg_write_scanlines(&cinfo, &row_pointer, 1);
    }

    // Termina la compressione
    jpeg_finish_compress(&cinfo);

    // Libera le risorse
    fclose(outfile);
    jpeg_destroy_compress(&cinfo);

    return 1;
}

// Convert an unsigned char image matrix to an float image matrix
void convertToFloat(const unsigned char *input, float *output, const int size)
{
    for (int i = 0; i < size; i++) {
        output[i] = (float)input[i];
    }
}

// Convert a float image matrix to unsigned char image matrix
void convertToUnsignedChar(const float *image_float, unsigned char *image_char, const int size) {
    for (int i = 0; i < size; i++) {
        image_char[i] = (unsigned char)fminf(fmaxf(image_float[i], 0.0f), 255.0f); // Clamp tra 0 e 255
        //image_char[i] = (unsigned char)image_float[i]; // Clamp tra 0 e 255
    }
}

/*quantization_matrix[8][8]
Id_x = gridId.x * blockDim.x + threadIdx.x
Id_y = gridID.y * blockDim.y + threadIdx.y
global = Id_y * gridDim.x * blockDim.x + Id_x
C[global] =  A[global] / quantization_matrix[threadIdx.x * BLOCK_SIZE + threadIdx.y]*/

// Kernel CUDA per la sottrazione element-wise matrice - scalare
__global__ void sub_matrix_scalar(const float* A, const float scalar, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] - scalar;
    }
}

// Kernel CUDA per l'addizione element-wise matrice - scalare
__global__ void add_matrix_scalar(const float* A, const float scalar, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] + scalar;
    }
}

// Kernel CUDA per la divisione elemento per elemento
__global__ void divide_matrices(const float* A, const float* B, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  round(A[global] / B[threadIdx.y * blockDim.x + threadIdx.x]);
    }
}

// Kernel CUDA per la moltiplicazione elemento per elemento
__global__ void multiply_matrices(const float* A, const float* B, float* C, const int size) {
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * blockDim.x + threadIdx.x;
    const int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int global = Id_y * gridDim.x * blockDim.x + Id_x;

    // Controlla che l'indice sia all'interno dei limiti
    if (global < size) {
        C[global] =  A[global] * B[threadIdx.y * blockDim.x + threadIdx.x];
    }
}

/* *
 * Effettua la DCT utilizzando la matrice di trasformazione
 * (TRASFORM_MATRIX @ IMAGE_MATRIX) @ TRANSFORM_MATRIX.T
 * La matrice di trasformazione è 8x8
 * result = TRASFORM_MATRIX @ IMAGE_MATRIX
 * result = result @ TRANSFORM_MATRIX.T
 * */
__global__ void cuda_matrix_dct(const float* image_matrix, const float* transform_matrix, float* result) {
    __shared__ float shared_matrix[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float shared_transform[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float shared_image[BLOCK_SIZE*BLOCK_SIZE];
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int global = Id_y * gridDim.x * BLOCK_SIZE + Id_x;

    float sums = 0;
    shared_transform[threadIdx.y * BLOCK_SIZE + threadIdx.x] = transform_matrix[threadIdx.y * BLOCK_SIZE + threadIdx.x];
    shared_image[threadIdx.y * BLOCK_SIZE + threadIdx.x] = image_matrix[global];
    __syncthreads();

    // result = transform_matrix @ image_matrix
    // result = transform_matrix[righe] @ image_matrix[colonne]
    for (int i = 0;i < BLOCK_SIZE;i++) {
        //sums += transform_matrix[threadIdx.y * BLOCK_SIZE + i] * image_matrix[(offset_y * gridDim.x * BLOCK_SIZE) + i * (gridDim.x * BLOCK_SIZE) + Id_x];
        //sums += shared_transform[threadIdx.y * BLOCK_SIZE + i] * image_matrix[(offset_y * gridDim.x * BLOCK_SIZE) + i * (gridDim.x * BLOCK_SIZE) + Id_x];
        sums += shared_transform[threadIdx.y * BLOCK_SIZE + i] * shared_image[threadIdx.x + i * BLOCK_SIZE];
    }
    // result[Id_y * gridDim.x * BLOCK_SIZE + Id_x] = sums;
    shared_matrix[threadIdx.y * BLOCK_SIZE + threadIdx.x] = sums;
    sums = 0;
    // Devo attendere il completamento della DOT precedente
    __syncthreads();

    // result = result(precedente) @ transform_matrix.T (trasposta)
    // result = result(precedente)[righe] @ transform_matrix[righe] (perchè la trasposta)
    for (int i = 0;i < BLOCK_SIZE;i++) {
        //sums += result[Id_y * (gridDim.x * BLOCK_SIZE) + offset_x + i] * transform_matrix[threadIdx.x * BLOCK_SIZE + i];
        sums += shared_matrix[threadIdx.y * BLOCK_SIZE + i] * shared_transform[threadIdx.x * BLOCK_SIZE + i];
    }
    result[Id_y * gridDim.x * BLOCK_SIZE + Id_x] = sums;
}

/* *
 * Effettua la IDCT utilizzando la matrice di trasformazione
 * (TRANSFORM_MATRIX.T @ DCT_MATRIX) @ TRANSFORM_MATRIX
 * La matrice di trasformazione è 8x8
 * result = TRANSFORM_MATRIX.T @ DCT_MATRIX
 * result = result @ TRANSFORM_MATRIX
 * */
__global__ void cuda_matrix_idct(const float* image_matrix, const float* transform_matrix, float* result) {
    __shared__ float shared_matrix[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float shared_transform[BLOCK_SIZE*BLOCK_SIZE];
    __shared__ float shared_image[BLOCK_SIZE*BLOCK_SIZE];
    // Calcola l'indice globale del thread
    const int Id_x = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    const int Id_y = blockIdx.y * blockDim.y + threadIdx.y;
    const int global = Id_y * gridDim.x * BLOCK_SIZE + Id_x;

    float sums = 0;
    shared_transform[threadIdx.y * BLOCK_SIZE + threadIdx.x] = transform_matrix[threadIdx.y * BLOCK_SIZE + threadIdx.x];
    shared_image[threadIdx.y * BLOCK_SIZE + threadIdx.x] = image_matrix[global];
    __syncthreads();

    // result = transform_matrix.T @ dct_matrix
    // result = transform_matrix[colonne](x trasposta) @ image_matrix[colonne]
    for (int i = 0;i < BLOCK_SIZE;i++) {
        //sums += shared_transform[i * BLOCK_SIZE + threadIdx.y] * image_matrix[(offset_y * gridDim.x * BLOCK_SIZE) + i * (gridDim.x * BLOCK_SIZE) + Id_x];
        sums += shared_transform[i * BLOCK_SIZE + threadIdx.y] * shared_image[threadIdx.x + i * BLOCK_SIZE];
    }
    shared_matrix[threadIdx.y * BLOCK_SIZE + threadIdx.x] = sums;
    sums = 0;
    __syncthreads();

    // result = result(precedente) @ transform_matrix
    // result = result[righe] @ transform_matrix[colonne]
    for (int i = 0;i < BLOCK_SIZE;i++) {
        sums += shared_matrix[threadIdx.y * BLOCK_SIZE + i] * shared_transform[i * BLOCK_SIZE + threadIdx.x];
    }
    result[Id_y * gridDim.x * BLOCK_SIZE + Id_x] = sums;
}

void dct_all_blocks_cuda(float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result)
{
    // Pre-alloca memoria GPU per i blocchi temporanei
    float *temp2,*d_Q_matrix;
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipGetSymbolAddress((void**)&d_Q_matrix,const_quant_matrix));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK SIZE
    const int gridx = img_width / BLOCK_SIZE;
    const int gridy = img_height / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(gridx, gridy);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // Lancio del kernel subsampling (--128)
    sub_matrix_scalar<<<gridDim, blockDim>>>(image_matrix, 128, image_matrix, img_width * img_height);

    // Lancio del kernel DCT
    cuda_matrix_dct<<<gridDim, blockDim>>>(image_matrix, transform_matrix, temp2);

    // Lancio del kernel quantizzazione
    divide_matrices<<<gridDim, blockDim>>>(temp2, d_Q_matrix, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("DCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp2));
}

void idct_all_blocks_cuda(const float* image_matrix, const int img_height, const int img_width, const float* transform_matrix, float* result)
{
    // Pre-alloca memoria GPU per i blocchi temporanei
    float* temp2, *d_Q_matrix;
    CHECK_CUDA(hipMalloc(&temp2, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipGetSymbolAddress((void**)&d_Q_matrix,const_quant_matrix));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    const int gridx = img_width / BLOCK_SIZE;
    const int gridy = img_height / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim(gridx, gridy);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // Lancio del kernel de-quantizzazione
    multiply_matrices<<<gridDim, blockDim>>>(image_matrix, d_Q_matrix, temp2, img_width * img_height);

    // Lancio del kernel IDCT
    cuda_matrix_idct<<<gridDim, blockDim>>>(temp2, transform_matrix, result);

    // Lancio del kernel inverse of subsampling (++128)
    add_matrix_scalar<<<gridDim, blockDim>>>(result, 128, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("IDCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp2));
}