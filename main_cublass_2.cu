//%%cuda --compiler-args "--library cublas --library jpeg -arch=sm_75"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"
#include "utils_kernels.cuh"
#include <hipblas.h>

// Check CUDA error
#define CHECK_CUDA(call)                                          \
    {                                                             \
        hipError_t err = call;                                   \
        if (err != hipSuccess)                                   \
        {                                                         \
            printf("%s : %d", hipGetErrorString(err), __LINE__); \
            exit(EXIT_FAILURE);                                   \
        }                                                         \
    }

// Check CUBLAS error
#define CHECK_CUBLAS(call)                                         \
    {                                                              \
        hipblasStatus_t err = call;                                 \
        if (err != HIPBLAS_STATUS_SUCCESS)                          \
        {                                                          \
            printf("CUBLAS error in %s : %d", __FILE__, __LINE__); \
            exit(EXIT_FAILURE);                                    \
        }                                                          \
    }

#define BLOCK_SIZE 8

__constant__ float const_quant_matrix[BLOCK_SIZE*BLOCK_SIZE];

// Using CUBLAS HANDLE to compute the DCT and the IDCT
__host__ void dct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle);
__host__ void idct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle);

int main()
{

    const char *filename = "camera256.tif.jpeg";
    int width, height, channels;

    /*// Load a jpeg image in image_matrix
    unsigned char *image_matrix = load_jpeg_as_matrix(filename, &width, &height, &channels);
    if (!image_matrix)
    {
        exit(EXIT_FAILURE);
    }

    // allocate host memory for the float image
    float *image_matrix_float;
    image_matrix_float = (float *)malloc(width * height * sizeof(float));
    convertToFloat(image_matrix, image_matrix_float, width * height * channels);
    free(image_matrix);*/

    width = 4096;
    height = 4096;

    float* image_matrix_float;
    image_matrix_float = (float*)malloc(width * height * sizeof(float));
    srand(41);
    for (int i = 0; i < height;i++) {
        for (int j = 0; j < width; j++) {
            image_matrix_float[i * width + j] = rand() % 256;;
        }
    }

    printf("Printing the 8x8 of image[] (matrix from the jpeg image w:%d h:%d)\n",width,height);
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", image_matrix_float[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Quantization matrix (su constant)
    float quant_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            16, 11, 10, 16, 24, 40, 51, 61,
            12, 12, 14, 19, 26, 58, 60, 55,
            14, 13, 16, 24, 40, 57, 69, 56,
            14, 17, 22, 29, 51, 87, 80, 62,
            18, 22, 37, 56, 68, 109, 103, 77,
            24, 35, 55, 64, 81, 104, 113, 92,
            49, 64, 78, 87, 103, 121, 120, 101,
            72, 92, 95, 98, 112, 100, 103, 99};
    CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(const_quant_matrix), quant_matrix, sizeof(quant_matrix)));

    // Transform matrix (hardcoded for simplicity)
    float transform_matrix[BLOCK_SIZE * BLOCK_SIZE] = {
            0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339, 0.35355339,
            0.5, 0.5, 0, 0, 0, 0, -0.5, -0.5,
            0.4472136, 0.2236068, -0.2236068, -0.4472136, -0.4472136, -0.2236068, 0.2236068, 0.4472136,
            0, 0, -0.70710678, 0, 0, 0.70710678, 0, 0,
            0.35355339, -0.35355339, -0.35355339, 0.35355339, 0.35355339, -0.35355339, -0.35355339, 0.35355339,
            0.5, -0.5, 0, 0, 0, 0, 0.5, -0.5,
            0.2236068, -0.4472136, 0.4472136, -0.2236068, -0.2236068, 0.4472136, -0.4472136, 0.2236068,
            0, 0, 0, -0.70710678, 0.70710678, 0, 0, 0};

    // Initialize CUBLAS
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    // allocate host memory for the result image
    float *result;
    result = (float *)malloc(width * height * sizeof(float));

    // allocate device memory for: image_block (d_A), trasform_matrix (d_B), and result (d_C)
    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, width * height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, BLOCK_SIZE * BLOCK_SIZE * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, width * height * sizeof(float)));

    // copy host memory to device
    CHECK_CUDA(hipMemcpy(d_A, image_matrix_float, width * height * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, transform_matrix, BLOCK_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyHostToDevice));

    // Compute DCT using CUBLAS
    // d_A = image_block | d_B = transform_matrix | d_C = result
    dct_all_blocks(d_A, height,width,d_B, d_C, handle);

    // copy result form device memory back to host
    // result = d_C
    CHECK_CUDA(hipMemcpy(result, d_C, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the dct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // d_E = result of the idct applied on the block_image compressed
    float *d_E;
    CHECK_CUDA(hipMalloc(&d_E, width * height * sizeof(float)));

    // Compute the idct
    // d_C = result ; d_B = transform_matrix ; d_E = result
    idct_all_blocks(d_C, height, width, d_B,d_E,handle);

    // copy device memory to host
    // result = d_E
    CHECK_CUDA(hipMemcpy(result, d_E, width * height * sizeof(float), hipMemcpyDeviceToHost));

    printf("Printing the 8x8 of result[] (matrix coming from the idct)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%f ", result[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    // Salva l'immagine in formato JPEG
    const char *filename_out = "output.jpg";
    int quality = 100; // Qualita JPEG (0-100)

    // allocate host memory for the usigned char image
    unsigned char *image_matrix_uc;
    image_matrix_uc = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    convertToUnsignedChar(result, image_matrix_uc, width * height);
    free(result);

    printf("Printing the 8x8 of U_C[] (unsignedchar)\n");
    for (int i = 0; i < BLOCK_SIZE; i++){
        for (int j = 0; j < BLOCK_SIZE; j++){
            printf("%d ", image_matrix_uc[i * width + j]);
        }
        printf("\n");
    }
    printf("\n\n");

    if (save_grayscale_jpeg(filename_out, image_matrix_uc, width, height, quality))
    {
        printf("Image saved successfully to %s\n", filename_out);
    }
    else
    {
        fprintf(stderr, "Error: Failed to save image\n");
    }


    // Cleanup
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_B));
    CHECK_CUDA(hipFree(d_C));
    CHECK_CUDA(hipFree(d_E));
    free(image_matrix_uc);
    free(image_matrix_float);
    return 0;
}

/* *
 * Funzione per l'applicazione della "compressione" (IN_IMAGE->scaling->dct->quantization->OUT_COMPRESSED)
 * L'utilizzo di una matrice di quantizzazione con tutti 1 ad esempio, ovviamente non porta a nessuna compressione
 * e/o perdita di qualita, in quando dct/idct sono reversibili a meno di piccoli errori di arrotondamento.
 * In questo caso particolare la matrice di quantizzazione e quella definita da JPEG (diversa da 1),
 * e quindi introduce compressione/perdita di qualita.
 * */
void dct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    // Pre-alloca memoria GPU per i blocchi temporanei
    float *temp1, *transform_matrix_expanded, *d_Q_matrix;
    CHECK_CUDA(hipMalloc(&temp1, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&transform_matrix_expanded, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipGetSymbolAddress((void**)&d_Q_matrix,const_quant_matrix));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE);
    dim3 gridDim(gridx,gridy);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // subsampling (--128)
    sub_matrix_scalar<<<gridDim,blockDim>>>(image_matrix, 128, image_matrix, img_width * img_height);

    // expand the T matrix to the same size of the image
    upgrade_T_matrix<<<gridDim,blockDim>>>(transform_matrix,transform_matrix_expanded,img_width * img_height);

    // Calcola temp1 = transform_matrix_expanded @ image_block
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, img_height, img_width, img_width,
                             &alpha, transform_matrix_expanded, img_height, image_matrix, img_width,
                             &beta, temp1, img_height));

    // Calcola temp1 @ transform_matrix_expanded.T
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, img_height, img_width, img_width,
                             &alpha, temp1, img_height, transform_matrix_expanded, img_width,
                             &beta, result, img_height));

    // Lancio del kernel quantizzazione
    divide_matrices<<<gridDim,blockDim>>>(result, d_Q_matrix, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("DCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);


    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp1));
    CHECK_CUDA(hipFree(transform_matrix_expanded));
}

/* *
 * Funzione per l'applicazione della "de-compressione" (IN_COMPRESSED->de-quantization->idct->UPscaling->OUT_IMAGE)
 * */
void idct_all_blocks(float *image_matrix, int img_height, int img_width, const float *transform_matrix, float *result, hipblasHandle_t handle)
{
    float alpha = 1.0f;
    float beta = 0.0f;

    // Pre-alloca memoria GPU per i blocchi temporanei
    float *temp1, *transform_matrix_expanded, *d_Q_matrix;
    CHECK_CUDA(hipMalloc(&temp1, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipMalloc(&transform_matrix_expanded, img_width * img_height * sizeof(float)));
    CHECK_CUDA(hipGetSymbolAddress((void**)&d_Q_matrix,const_quant_matrix));

    // Configurazione della griglia e dei blocchi
    // -> using BLOCK_SIZE
    int gridx = img_width / BLOCK_SIZE;
    int gridy = img_width / BLOCK_SIZE;
    dim3 blockDim(BLOCK_SIZE,BLOCK_SIZE);
    dim3 gridDim(gridx,gridy);

    // Avvia il timer
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    CHECK_CUDA(hipEventRecord(start, 0));

    // Lancio del kernel de-quantizzazione
    multiply_matrices<<<gridDim,blockDim>>>(image_matrix, d_Q_matrix, image_matrix, img_width * img_height);

    // expand the T matrix to the same size of the image
    upgrade_T_matrix<<<gridDim,blockDim>>>(transform_matrix,transform_matrix_expanded,img_width * img_height);

    // Compute temp1 = transform_matrix.T @ image_block
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, img_height, img_width, img_width,
                             &alpha, transform_matrix_expanded, img_height, image_matrix, img_width,
                             &beta, temp1, img_height));

    // Compute temp1 @ transform_matrix
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, img_height, img_width, img_width,
                             &alpha, temp1, img_height, transform_matrix_expanded, img_width,
                             &beta, result, img_height));

    // inverse of subsampling (++128)
    add_matrix_scalar<<<gridDim,blockDim>>>(result, 128, result, img_width * img_height);

    // Ferma il timer
    CHECK_CUDA(hipEventRecord(stop, 0));
    CHECK_CUDA(hipEventSynchronize(stop));
    // Calcola il tempo totale
    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("IDCT (%d,%d): %f ms\n",img_width,img_height,milliseconds);

    // Libera memoria GPU
    CHECK_CUDA(hipFree(temp1));
    CHECK_CUDA(hipFree(transform_matrix_expanded));
}
